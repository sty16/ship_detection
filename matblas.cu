#include "hip/hip_runtime.h"
#include"matblas.h"

// 使用显存前必须初始化，因为该显存可能已经使用过
__global__ void cuMatMul(cuMat a, cuMat b, cuMat res, hipComplex alpha) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;
    if( i < a.height && j < b.width){
        res.data[i][j] = make_hipComplex(0.0, 0.0);
        for(int k=0;k<a.width;k++)
        {
            res.data[i][j] = hipCaddf(res.data[i][j],hipCmulf(a.data[i][k],b.data[k][j]));
        }
        res.data[i][j] = hipCmulf(alpha, res.data[i][j]);
    }
    // hipCaddf add two hipComplex; hipCmulf multiply two hipComplex
}

__global__ void cuMatHer(cuMat a, cuMat res, hipComplex alpha){
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;
    if(i < res.height && j < res.width){
        res.meta_data[INDEX(i, j, res.width)] = make_hipComplex(0, 0);
        for(int k = 0;k < a.width; k++){
            res.meta_data[INDEX(i, j, res.width)] = hipCaddf(res.meta_data[INDEX(i, j, res.width)], hipCmulf(a.meta_data[INDEX(i, k, a.width)], hipConjf(a.meta_data[INDEX(j, k, a.width)])));    
        }
        res.meta_data[INDEX(i, j, res.width)] = hipCmulf(alpha, res.meta_data[INDEX(i, j, res.width)]);
    }
}

__global__ void  cuMatPad(cuMat a, cuMat res, int pad_row, int pad_col){
    int i = threadIdx.x + blockDim.x * blockIdx.x;  // the ith row
    int j = threadIdx.y + blockDim.y * blockIdx.y;
    if(i<res.height && j < res.width){
        if(i<pad_row){
            if(j<pad_col){
                res.data[i][j] = a.data[pad_row - 1 - i][pad_col - 1 -j];
            }
            else if(j >= pad_col && j<pad_col+a.width){
                res.data[i][j] = a.data[pad_row - 1 -i][j-pad_col];
            }
            else{
                res.data[i][j] = a.data[pad_row - 1 - i][2*a.width+pad_col-1-j];
            }
        }else if(i < pad_row + a.height && i >= pad_row){
            if(j<pad_col){
                res.data[i][j] = a.data[i-pad_row][pad_col - 1 -j];
            }
            else if(j >= pad_col && j<pad_col+a.width){
                res.data[i][j] = a.data[i-pad_row][j-pad_col];
            }
            else{
                res.data[i][j] = a.data[i-pad_row][2*a.width+pad_col-1-j];
            }
        }else{
            if(j<pad_col){
                res.data[i][j] = a.data[2*a.height+pad_row-1-i][pad_col - 1 -j];
            }
            else if(j >= pad_col && j<pad_col+a.width){
                res.data[i][j] = a.data[2*a.height+pad_row-1-i][j-pad_col];
            }
            else{
                res.data[i][j] = a.data[2*a.height+pad_row-1-i][2*a.width+pad_col-1-j];
            }
       }
    }
}

__global__ void transposeSmem(cuMat a, cuMat res){                  // use shared memory to transpose matrix
    int i = threadIdx.x + blockDim.x * blockIdx.x;  
    int j = threadIdx.y + blockDim.y * blockIdx.y;
    __shared__ hipComplex tile[BDIMX][BDIMY];
    int row, col, trow, tcol;
    int m = a.height/blockDim.x;        // the number of full filled block
    int n = a.width/blockDim.y;
    if(blockIdx.x < m && blockIdx.y < n)
    {                                                                              // full block and non-full block
        tile[threadIdx.x][threadIdx.y] = a.data[i][j]; 
        int numx;                       // find the index  
        numx = threadIdx.x*blockDim.y + threadIdx.y;
        trow = numx / blockDim.x;
        tcol = numx % blockDim.x;
        row = trow + blockIdx.y*blockDim.y;
        col = tcol + blockIdx.x*blockDim.x;
    }else{
        row = j;col = i;
    }
   __syncthreads();                                                                  //wait for the tile filled with value;
    if(row<res.height && col<res.width){
        if(blockIdx.x < m && blockIdx.y < n){
            res.data[row][col] = hipConjf(tile[tcol][trow]);                                        //coalesced  write
        }else{
            res.data[row][col] = hipConjf(a.data[i][j]);
        }
    }
}

__global__ void transposeDmem(cuMat a, cuMat res){
    int i = threadIdx.x + blockDim.x * blockIdx.x;  
    int j = threadIdx.y + blockDim.y * blockIdx.y;
    __shared__ hipComplex tile[BDIMX][BDIMY];
    int row, col, trow, tcol;
    int m = a.height/blockDim.x;        // the number of full filled block
    int n = a.width/blockDim.y;
    if(blockIdx.x < m && blockIdx.y < n)
    {                                                                              // full block and non-full block
        tile[threadIdx.x][threadIdx.y] = a.meta_data[INDEX(i, j, a.width)]; 
        int numx;                       // find the index  
        numx = threadIdx.x*blockDim.y + threadIdx.y;
        trow = numx / blockDim.x;
        tcol = numx % blockDim.x;
        row = trow + blockIdx.y*blockDim.y;
        col = tcol + blockIdx.x*blockDim.x;
    }else{
        row = j;col = i;
    }
   __syncthreads();                                                                  //wait for the tile filled with value;
    if(row<res.height && col<res.width){
        if(blockIdx.x < m && blockIdx.y < n){
            res.meta_data[INDEX(row, col, res.width)] = hipConjf(tile[tcol][trow]);                                        //coalesced  write
        }else{
            res.meta_data[INDEX(row, col, res.width)] = hipConjf(a.meta_data[INDEX(i, j, a.width)]);
        }
    }
}

__global__ void cuMatInv(cuMat a, cuMat res, hipComplex det, char *begin, int threadsize)
{
    // detsize 每个线程的字节数
    int i = threadIdx.x + blockDim.x * blockIdx.x;  
    int j = threadIdx.y + blockDim.y * blockIdx.y;
    int pointer = 0;
    if(i < a.height && j < a.width)
    {
        char *threadMempool = (char *)begin + INDEX(i, j, a.width)*threadsize;
        cuMat temp;
        DeviceInitMat(temp, threadMempool, pointer, a.height-1, a.width-1);
        for(int k = 0;k < temp.height; k++)
        {
            for(int t = 0; t < temp.width; t++)
            {
                int row = k>=i?k+1:k;
                int col = t>=j?t+1:t;
                temp.meta_data[INDEX(k, t, temp.width)] = a.meta_data[INDEX(row, col, a.width)];
            }
        }
        hipComplex  temp_det = MatDet(temp, threadMempool, pointer);
        res.meta_data[INDEX(j, i, res.width)] = hipCdivf(temp_det, det);
        if((i+j)%2 == 1)
        {
            hipComplex alpha = make_hipComplex(-1, 0);
            res.meta_data[INDEX(j, i, res.width)] = hipCmulf(res.meta_data[INDEX(j, i, res.width)], alpha);
        }
    }
}

__device__ void InitMat(cuMat &mat, int h, int w){
    mat.height = h;
    mat.width = w;
    hipMalloc((void**)&mat.data, sizeof(hipComplex *)*h);
    hipMalloc((void**)&mat.meta_data, sizeof(hipComplex)*h*w);
    for(int i=0;i<h;i++){
        mat.data[i] = mat.meta_data + i*w;
    }
}

__host__ void HostInitMat(cuMat &mat, int h, int w){
    mat.height = h;
    mat.width = w;
    hipMallocManaged((void**)&mat.data, sizeof(hipComplex *)*h);
    hipMallocPitch((void**)&mat.meta_data, &mat.pitch ,sizeof(hipComplex)*w, h);   //采用hipMallocPitch分配2D数组加快访问
    for(size_t i=0;i<h;i++)
    {
        mat.data[i] =  (hipComplex *)((char *)mat.meta_data + i*mat.pitch);     //直接访问设备内存会报错，使用hipMallocManaged
    }
    // Memory does not need to be initialized to ensure speed
}

__device__ void DeviceInitMat(cuMat &mat, char *begin, int &pointer, int h, int w){
    // begin 线程数据起使地址 pointer当前的指针字节位置
    mat.meta_data = (hipComplex *)((char *)begin + pointer); //分配矩阵地址
    pointer = pointer + h*w*sizeof(hipComplex);    // 指针进行偏移
    if(pointer >=  THREADSPACE){
        printf("ErrorMallocAllocation\n");
        mat.height = 0;
        mat.width = 0;
        mat.meta_data = (hipComplex *)begin;      //回到起始空间
    }else{
        mat.height = h;
        mat.width = w;
    }
}

__host__ void HostInitImg(cuImg &img, int h, int w){
    img.height = h;
    img.width = w;
    hipMallocManaged((void **)&img.data, sizeof(uint8 *)*h);
    hipMallocPitch((void **)&img.meta_data, &img.pitch, sizeof(uint8)*w, h);
    for(size_t i = 0;i<h;i++)
    {
        img.data[i] = (uint8 *)((char *)img.meta_data + i*img.pitch);
    }
}

__device__ void DestroyMat(cuMat &mat){
    hipFree(mat.data);
    hipFree(mat.meta_data);
}

__host__ void HostDestroyMat(cuMat &mat){
    hipFree(mat.data);
    hipFree(mat.meta_data);
}

__host__ void HostDestroyImg(cuImg &img){
    hipFree(img.data);
    hipFree(img.meta_data);
}

__device__ void DeviceDestroyMat(cuMat mat, char *begin, int &pointer)
{
    pointer = pointer - mat.height*mat.width*sizeof(hipComplex);
    hipComplex *temp = (hipComplex *)((char *)begin + pointer);
    if(temp != mat.meta_data)
    {
        printf("cudaFreeFailure\n");
        pointer = pointer + mat.height*mat.width*sizeof(hipComplex);   //线程空间相当于栈空间，注意先进后出的释放顺序
    }
}

__device__  cuMat  MulMat(cuMat a, cuMat b, hipComplex alpha){
   cuMat res; 
   if(a.width == b.height){
        InitMat(res, a.height, b.width);
        dim3 blockdim(16, 16);
        dim3 griddim(a.height/16 + 1, b.width/16 + 1);
        cuMatMul<<<griddim,blockdim>>>(a, b, res, alpha);    // Compute matrix multiplication in parallel
        hipDeviceSynchronize();   // parent kernel waits for child kernel 
        return res;
   }else{
        printf("the size of two input Matrix are not match\n");
        InitMat(res, 1, 1);
        return res;
   }
}

__host__ cuMat HostPadMat(cuMat a, int pad_row, int pad_col){
    cuMat res;
    HostInitMat(res, a.height + 2*pad_row, a.width + 2*pad_col);   // 主机调用, 分配设备内存
    dim3 blockdim(32, 32);
    dim3 griddim((int)(res.height/32 + 1), (int)(res.width/32 + 1));                         // pad by row
    cuMatPad<<<griddim,blockdim>>>(a, res, pad_row, pad_col);
    hipDeviceSynchronize(); 
    return res;
}

__device__ cuMat TransposeMat(cuMat a){
    cuMat res;
    InitMat(res, a.width, a.height);
    dim3 blockdim(32, 32);
    dim3 griddim(a.height/32 + 1, a.width/32 + 1);
    transposeSmem<<<griddim, blockdim>>>(a, res); 
    hipDeviceSynchronize();
    printf("%d", res.height);
    return res;
} 

__device__ cuMat DeviceTransMat(cuMat a, char *begin, int &pointer){
    cuMat res;
    DeviceInitMat(res, begin, pointer, a.width, a.height);
    dim3 blockdim(32, 32);
    dim3 griddim(a.height/32 + 1, a.width/32 + 1);
    transposeSmem<<<griddim, blockdim>>>(a, res); 
    hipDeviceSynchronize();
    printf("%d", res.height);
    return res;
} 

__device__ cuMat HerMat(cuMat mat, char *begin, int &pointer, hipComplex alpha)
{
    cuMat res;
    DeviceInitMat(res, begin, pointer, mat.height, mat.height);
    dim3 blockdim(16, 16);
    dim3 griddim(res.height/16 + 1, res.width/16 + 1);
    cuMatHer<<<griddim, blockdim>>>(mat, res, alpha);
    hipDeviceSynchronize();
    return res;
}

__device__ hipComplex MatDet(cuMat mat, char *begin, int &pointer)
{
    if(mat.height != mat.width)
    {
        printf("the height and width of the matrix are not match\n");
        return make_hipComplex(0, 0);
    }
    if(mat.height == 1)
    {
        hipComplex det = mat.meta_data[INDEX(0, 0, mat.width)];
        return det;
    }
    if(mat.height == 2){
        hipComplex det = hipCsubf(hipCmulf(mat.meta_data[0], mat.meta_data[3]), hipCmulf(mat.meta_data[1], mat.meta_data[2]));
        return det;
    }
    cuMat temp;
    DeviceInitMat(temp, begin, pointer, mat.height-1, mat.width-1);
    hipComplex det = make_hipComplex(0, 0);
    int row, col;
    for(int i = 0; i < mat.height; i++)
    {
        for(int j = 0; j < mat.height - 1; j++)
        {
            for(int k = 0; k < mat.width - 1; k++)
            {
                row = j + 1;
                col = (k>=i)?k+1:k;
                temp.meta_data[INDEX(j, k, temp.width)] = mat.meta_data[INDEX(row, col, mat.width)];
            }
        }
        hipComplex cofactor = MatDet(temp, begin, pointer);
        // printf("%f,%f\n", cofactor.x, cofactor.y);
        if(i%2 == 0)
        {
            det = hipCaddf(det, hipCmulf(mat.meta_data[INDEX(0, i, mat.width)], cofactor));
        }else{
            det = hipCsubf(det, hipCmulf(mat.meta_data[INDEX(0, i, mat.width)], cofactor));
        }
    }
    DeviceDestroyMat(temp, begin, pointer);    // 释放递归过程中分配的线程栈空间
    return det;
}

__device__ hipComplex ComputeDet(cuMat mat){
    // only for matrix with size 3x3
    hipComplex res = make_hipComplex(0,0);
    hipComplex det1 = hipCsubf(hipCmulf(mat.meta_data[4], mat.meta_data[8]), hipCmulf(mat.meta_data[5], mat.meta_data[7]));
    hipComplex det2 = hipCsubf(hipCmulf(mat.meta_data[3], mat.meta_data[8]), hipCmulf(mat.meta_data[5], mat.meta_data[6]));
    hipComplex det3 = hipCsubf(hipCmulf(mat.meta_data[3], mat.meta_data[7]), hipCmulf(mat.meta_data[4], mat.meta_data[6]));
    res = hipCaddf(res, hipCmulf(mat.meta_data[0], det1));
    res = hipCsubf(res, hipCmulf(mat.meta_data[1], det2));
    res = hipCaddf(res, hipCmulf(mat.meta_data[2], det3));
    return res;
}

__device__ cuMat MatInv(cuMat mat, char *begin, int &pointer)
{
    cuMat res, temp;
    if(mat.height != mat.width)
    {
        printf("The matrix is not a square matrix\n");
        res.height = 0; res.width = 0;
        return res;
    }else{
        hipComplex mat_det = ComputeDet(mat);
        mat_det = MatDet(mat, begin, pointer);
        if(hipCabsf(mat_det) < (float) 1e-5)
        {
            printf("the matrix is strange");          //矩阵奇异
            res.height = 0;res.width = 0;
            return res;
        }
        DeviceInitMat(res, begin, pointer, mat.height, mat.width); 
        if(mat.height == 1)
        {
            hipComplex temp = make_hipComplex(1, 0);
            res.meta_data[0] = hipCdivf(temp, mat_det);
            return res;
        }
        DeviceInitMat(temp, begin, pointer, res.height-1, res.width-1);
        for(int i=0; i<res.height;i++)
        {
            for(int j=0;j<res.width;j++)
            {
                for(int k=0;k<temp.height;k++)
                {
                    for(int t=0;t<temp.width;t++)
                    {
                        int row = k>=i?k+1:k;
                        int col = t>=j?t+1:t;
                        temp.meta_data[INDEX(k, t, temp.width)] = mat.meta_data[INDEX(row, col, mat.width)];
                    }
                } 
                hipComplex temp_det = hipCsubf(hipCmulf(temp.meta_data[0],temp.meta_data[3]), hipCmulf(temp.meta_data[1], temp.meta_data[2]));
                // temp_det = MatDet(temp, begin, pointer);
                // printf("%f,%f\n", temp_det.x, temp_det.y);
                res.meta_data[INDEX(j, i, res.width)] = hipCdivf(temp_det, mat_det);
                if((i+j)%2 == 1)
                {
                    hipComplex alpha = make_hipComplex(-1, 0);
                    res.meta_data[INDEX(j, i, res.width)] = hipCmulf(res.meta_data[INDEX(j, i, res.width)], alpha);
                }
            }
        }
        DeviceDestroyMat(temp, begin, pointer);
        return res;
    }
}// 采用并行方式快速求逆矩阵    一个kernel调用完成一行的元素求逆矩阵，还是在线程的空间进行分配 给出去当前的地址，作为新kernel的起始地址，并判断
//是否还有足够的空间，此外还可以通过共享内存
__device__ cuMat MatInvParal(cuMat mat, char *begin, int &pointer)
{
    // 计算并行所需的内存空间看是否还能够满足 can approve the max size smaller than 32x32
    cuMat res;
    if(mat.height != mat.width)
    {
        printf("The matrix is not a square matrix\n");
        res.height = 0; res.width = 0;
        return res;
    }else{
        hipComplex mat_det;
        mat_det = MatDet(mat, begin, pointer);
        if(hipCabsf(mat_det) < (float) 1e-5)
        {
            printf("the matrix is strange");          //矩阵奇异
            res.height = 0;res.width = 0;
            return res;
        }
        DeviceInitMat(res, begin, pointer, mat.height, mat.width); 
        if(mat.height == 1)
        {
            hipComplex temp = make_hipComplex(1, 0);
            res.meta_data[0] = hipCdivf(temp, mat_det);
            return res;
        }
        int threadsize = ((mat.height-1)*(mat.width-1) + (mat.width-1)*(mat.width)*(2*mat.width-1)/6 + 50)*sizeof(hipComplex);
        int InvSize = mat.height*mat.width*threadsize;  // 50 to insure enough space
        if((pointer + InvSize) > THREADSPACE){
            printf("the matrix is too large and can be inversed in the threadspace\n");
            DeviceDestroyMat(res, begin, pointer);
            res.height = 0;res.width = 0;
            return res;
        }else{
            char *InvStart = (char *)begin + pointer;
            dim3 blockdim(32, 32);
            dim3 griddim(1, 1);
            cuMatInv<<<griddim, blockdim>>>(mat, res, mat_det, InvStart, threadsize);
            hipDeviceSynchronize();
            return res;
        }
    }
}