#include "hip/hip_runtime.h"
#include"matblas.h"

__global__ void cuMatMul(cuMat a, cuMat b, cuMat res, hipComplex alpha) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;
    if( i < a.height && j < b.width){
        res.data[i][j] = make_hipComplex(0.0, 0.0);
        for(int k=0;k<a.width;k++)
        {
            res.data[i][j] = hipCaddf(res.data[i][j],hipCmulf(a.data[i][k],b.data[k][j]));
        }
        res.data[i][j] = hipCmulf(alpha, res.data[i][j]);
    }
    // hipCaddf add two hipComplex; hipCmulf multiply two hipComplex
}

__global__ void cuMatHer(cuMat a, cuMat res, hipComplex alpha){
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;
    if(i < res.height && j < res.width){
        res.meta_data[INDEX(i, j, res.width)] = make_hipComplex(0, 0);
        for(int k = 0;k < a.width; k++){
            res.meta_data[INDEX(i, j, res.width)] = hipCaddf(res.meta_data[INDEX(i, j, res.width)], hipCmulf(a.meta_data[INDEX(i, k, a.width)], hipConjf(a.meta_data[INDEX(j, k, a.width)])));    
        }
        res.meta_data[INDEX(i, j, res.width)] = hipCmulf(alpha, res.meta_data[INDEX(i, j, res.width)]);
    }
}

__global__ void  cuMatPad(cuMat a, cuMat res, int pad_row, int pad_col){
    int i = threadIdx.x + blockDim.x * blockIdx.x;  // the ith row
    int j = threadIdx.y + blockDim.y * blockIdx.y;
    if(i<res.height && j < res.width){
        if(i<pad_row){
            if(j<pad_col){
                res.data[i][j] = a.data[pad_row - 1 - i][pad_col - 1 -j];
            }
            else if(j >= pad_col && j<pad_col+a.width){
                res.data[i][j] = a.data[pad_row - 1 -i][j-pad_col];
            }
            else{
                res.data[i][j] = a.data[pad_row - 1 - i][2*a.width+pad_col-1-j];
            }
        }else if(i < pad_row + a.height && i >= pad_row){
            if(j<pad_col){
                res.data[i][j] = a.data[i-pad_row][pad_col - 1 -j];
            }
            else if(j >= pad_col && j<pad_col+a.width){
                res.data[i][j] = a.data[i-pad_row][j-pad_col];
            }
            else{
                res.data[i][j] = a.data[i-pad_row][2*a.width+pad_col-1-j];
            }
        }else{
            if(j<pad_col){
                res.data[i][j] = a.data[2*a.height+pad_row-1-i][pad_col - 1 -j];
            }
            else if(j >= pad_col && j<pad_col+a.width){
                res.data[i][j] = a.data[2*a.height+pad_row-1-i][j-pad_col];
            }
            else{
                res.data[i][j] = a.data[2*a.height+pad_row-1-i][2*a.width+pad_col-1-j];
            }
       }
    }
}

__global__ void transposeSmem(cuMat a, cuMat res){                  // use shared memory to transpose matrix
    int i = threadIdx.x + blockDim.x * blockIdx.x;  
    int j = threadIdx.y + blockDim.y * blockIdx.y;
    __shared__ hipComplex tile[BDIMX][BDIMY];
    int row, col, trow, tcol;
    int m = a.height/blockDim.x;        // the number of full filled block
    int n = a.width/blockDim.y;
    if(blockIdx.x < m && blockIdx.y < n)
    {                                                                              // full block and non-full block
        tile[threadIdx.x][threadIdx.y] = a.data[i][j]; 
        int numx;                       // find the index  
        numx = threadIdx.x*blockDim.y + threadIdx.y;
        trow = numx / blockDim.x;
        tcol = numx % blockDim.x;
        row = trow + blockIdx.y*blockDim.y;
        col = tcol + blockIdx.x*blockDim.x;
    }else{
        row = j;col = i;
    }
   __syncthreads();                                                                  //wait for the tile filled with value;
    if(row<res.height && col<res.width){
        if(blockIdx.x < m && blockIdx.y < n){
            res.data[row][col] = hipConjf(tile[tcol][trow]);                                        //coalesced  write
        }else{
            res.data[row][col] = hipConjf(a.data[i][j]);
        }
    }
}

__global__ void transposeDmem(cuMat a, cuMat res){
    int i = threadIdx.x + blockDim.x * blockIdx.x;  
    int j = threadIdx.y + blockDim.y * blockIdx.y;
    __shared__ hipComplex tile[BDIMX][BDIMY];
    int row, col, trow, tcol;
    int m = a.height/blockDim.x;        // the number of full filled block
    int n = a.width/blockDim.y;
    if(blockIdx.x < m && blockIdx.y < n)
    {                                                                              // full block and non-full block
        tile[threadIdx.x][threadIdx.y] = a.meta_data[INDEX(i, j, a.width)]; 
        int numx;                       // find the index  
        numx = threadIdx.x*blockDim.y + threadIdx.y;
        trow = numx / blockDim.x;
        tcol = numx % blockDim.x;
        row = trow + blockIdx.y*blockDim.y;
        col = tcol + blockIdx.x*blockDim.x;
    }else{
        row = j;col = i;
    }
   __syncthreads();                                                                  //wait for the tile filled with value;
    if(row<res.height && col<res.width){
        if(blockIdx.x < m && blockIdx.y < n){
            res.meta_data[INDEX(row, col, res.width)] = hipConjf(tile[tcol][trow]);                                        //coalesced  write
        }else{
            res.meta_data[INDEX(row, col, res.width)] = hipConjf(a.meta_data[INDEX(i, j, a.width)]);
        }
    }
}

__device__ void InitMat(cuMat &mat, int h, int w){
    mat.height = h;
    mat.width = w;
    hipMalloc((void**)&mat.data, sizeof(hipComplex *)*h);
    hipMalloc((void**)&mat.meta_data, sizeof(hipComplex)*h*w);
    for(int i=0;i<h;i++){
        mat.data[i] = mat.meta_data + i*w;
    }
}

__host__ void HostInitMat(cuMat &mat, int h, int w){
    mat.height = h;
    mat.width = w;
    hipMallocManaged((void**)&mat.data, sizeof(hipComplex *)*h);
    hipMallocPitch((void**)&mat.meta_data, &mat.pitch ,sizeof(hipComplex)*w, h);   //采用hipMallocPitch分配2D数组加快访问
    for(size_t i=0;i<h;i++)
    {
        mat.data[i] =  (hipComplex *)((char *)mat.meta_data + i*mat.pitch);     //直接访问设备内存会报错，使用hipMallocManaged
    }
    // Memory does not need to be initialized to ensure speed
}

__device__ void DeviceInitMat(cuMat &mat, char *begin, int &pointer, int h, int w){
    // begin 线程数据起使地址 pointer当前的指针字节位置
    mat.meta_data = (hipComplex *)((char *)begin + pointer); //分配矩阵地址
    pointer = pointer + h*w*sizeof(hipComplex);    // 指针进行偏移
    if(pointer >=  THREADSPACE){
        printf("ErrorMallocAllocation\n");
        mat.height = 0;
        mat.width = 0;
        mat.meta_data = (hipComplex *)begin;      //回到起始空间
    }else{
        mat.height = h;
        mat.width = w;
    }
}

__host__ void HostInitImg(cuImg &img, int h, int w){
    img.height = h;
    img.width = w;
    hipMallocManaged((void **)&img.data, sizeof(uint8 *)*h);
    hipMallocPitch((void **)&img.meta_data, &img.pitch, sizeof(uint8)*w, h);
    for(size_t i = 0;i<h;i++)
    {
        img.data[i] = (uint8 *)((char *)img.meta_data + i*img.pitch);
    }
}

__device__ void DestroyMat(cuMat &mat){
    hipFree(mat.data);
    hipFree(mat.meta_data);
}

__host__ void HostDestroyMat(cuMat &mat){
    hipFree(mat.data);
    hipFree(mat.meta_data);
}

__host__ void HostDestroyImg(cuImg &img){
    hipFree(img.data);
    hipFree(img.meta_data);
}

__device__ void DeviceDestroyMat(cuMat mat, char *begin, int &pointer)
{
    pointer = pointer - mat.height*mat.width*sizeof(hipComplex);
    hipComplex *temp = (hipComplex *)((char *)begin + pointer);
    if(temp != mat.meta_data)
    {
        printf("cudaFreeFailure\n");
        pointer = pointer + mat.height*mat.width*sizeof(hipComplex);   //线程空间相当于栈空间，注意先进后出的释放顺序
    }
}

__device__  cuMat  MulMat(cuMat a, cuMat b, hipComplex alpha){
   cuMat res; 
   if(a.width == b.height){
        InitMat(res, a.height, b.width);
        dim3 blockdim(16, 16);
        dim3 griddim(a.height/16 + 1, b.width/16 + 1);
        cuMatMul<<<griddim,blockdim>>>(a, b, res, alpha);    // Compute matrix multiplication in parallel
        hipDeviceSynchronize();   // parent kernel waits for child kernel 
        return res;
   }else{
        printf("the size of two input Matrix are not match\n");
        InitMat(res, 1, 1);
        return res;
   }
}

__host__ cuMat HostPadMat(cuMat a, int pad_row, int pad_col){
    cuMat res;
    HostInitMat(res, a.height + 2*pad_row, a.width + 2*pad_col);   // 主机调用, 分配设备内存
    dim3 blockdim(32, 32);
    dim3 griddim((int)(res.height/32 + 1), (int)(res.width/32 + 1));                         // pad by row
    cuMatPad<<<griddim,blockdim>>>(a, res, pad_row, pad_col);
    hipDeviceSynchronize(); 
    return res;
}

__device__ cuMat TransposeMat(cuMat a){
    cuMat res;
    InitMat(res, a.width, a.height);
    dim3 blockdim(32, 32);
    dim3 griddim(a.height/32 + 1, a.width/32 + 1);
    transposeSmem<<<griddim, blockdim>>>(a, res); 
    hipDeviceSynchronize();
    printf("%d", res.height);
    return res;
} 

__device__ cuMat DeviceTransMat(cuMat a, char *begin, int &pointer){
    cuMat res;
    DeviceInitMat(res, begin, pointer, a.width, a.height);
    dim3 blockdim(32, 32);
    dim3 griddim(a.height/32 + 1, a.width/32 + 1);
    transposeSmem<<<griddim, blockdim>>>(a, res); 
    hipDeviceSynchronize();
    printf("%d", res.height);
    return res;
} 

__device__ cuMat HerMat(cuMat mat, char *begin, int &pointer, hipComplex alpha)
{
    cuMat res;
    DeviceInitMat(res, begin, pointer, mat.height, mat.height);
    dim3 blockdim(16, 16);
    dim3 griddim(res.height/16 + 1, res.width/16 + 1);
    cuMatHer<<<griddim, blockdim>>>(mat, res, alpha);
    hipDeviceSynchronize();
    return res;
}

__device__ hipComplex MatDet(cuMat mat, char *begin, int &pointer)
{
    if(mat.height != mat.width)
    {
        printf("the height and width of the matrix are not match\n");
        return make_hipComplex(0, 0);
    }
    if(mat.height == 1)
    {
        hipComplex det = mat.meta_data[INDEX(0, 0, mat.width)];
        return det;
    }
    cuMat temp;
    DeviceInitMat(temp, begin, pointer, mat.height-1, mat.width-1);
    hipComplex det = make_hipComplex(0, 0);
    int row, col;
    for(int i = 0; i < mat.height; i++)
    {
        for(int j = 0; j < mat.height - 1; j++)
        {
            for(int k = 0; k < mat.width - 1; k++)
            {
                row = j + 1;
                col = (k>=i)?k+1:k;
                temp.meta_data[INDEX(j, k, temp.width)] = mat.meta_data[INDEX(row, col, mat.width)];
            }
        }
        hipComplex cofactor = MatDet(temp, begin, pointer);
        // printf("%f,%f\n", cofactor.x, cofactor.y);
        if(i%2 == 0)
        {
            det = hipCaddf(det, hipCmulf(mat.meta_data[INDEX(0, i, mat.width)], cofactor));
        }else{
            det = hipCsubf(det, hipCmulf(mat.meta_data[INDEX(0, i, mat.width)], cofactor));
        }
    }
    DeviceDestroyMat(temp, begin, pointer);    // 释放递归过程中分配的线程栈空间
    return det;
}