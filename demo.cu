#include "hip/hip_runtime.h"
#include"mat_read.h"
#include"matblas.h"
#include<stdio.h>
#include""
#include<hip/hip_runtime.h>
#include<hipblas.h>
#include<hip/hip_runtime_api.h>
#include<helper_functions.h>

//global variable
__device__ __managed__ cuMat imgPad[3]; //device variable managed variable,to which Host is accessible
__device__ __managed__ cuMat img[3];  // the matrix of the origin image

__global__ void PrintData() 
{
    printf("(%f,%f)\n", imgPad[0].data[1015][1015].x, imgPad[0].data[1015][1015].y);
}

__global__ void test() {
    hipFloatComplex *a;
    hipMalloc((void**)&a, sizeof(hipFloatComplex)*1000);
    // use hipComplex     
    cuMat A, B, C, D;
    hipComplex alpha = make_hipComplex(2.0, 0.0);
    InitMat(A, 6, 6);
    InitMat(B, 6, 6);
    for(int i=0;i<6;i++)
    {
        for(int j=0;j<6;j++)
        {
            A.data[i][j] = make_hipComplex(i,j);
            B.data[i][j] = make_hipComplex(i,j);
        }
    }
    C = MulMat(A, B, alpha);
    __syncthreads();
    // D = PadMat(A, 2, 2);
    __syncthreads();
    DestroyMat(A);
    DestroyMat(B);
    // double *alpha, *beta;
    // alpha = new double(1);
    // beta = new double(0);
    // hipblasHandle_t handle;
    // // hipblasCreate(&handle);
    // hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 3, 3, 2, alpha, d_B, 2, d_B, 2, beta, d_C, 3);
    // printf("%f\n", d_C[1]); 
}


int main(){
    const char *matfile_HH = "./data/imagery_HH.mat";
    const char *param_HH = "imagery_HH";
    const char *matfile_HV = "./data/imagery_HV.mat";
    const char *param_HV = "imagery_HV";
    const char *matfile_VV = "./data/imagery_VV.mat";
    const char *param_VV = "imagery_VV";
    complex<float> *img_HH, *img_HV, *img_VV;
    int h = 1000, w = 1000, N = 15;                               //size of the image data
    img_HH = matToArray(matfile_HH, param_HH);
    img_HV = matToArray(matfile_HV, param_HV);
    img_VV = matToArray(matfile_VV, param_VV);
    for(int i=0;i<3;i++){
        HostInitMat(img[i], h, w);
        hipMemcpy2D(img[i].meta_data, img[i].pitch, img_HH, sizeof(hipComplex)*w, sizeof(hipComplex)*w, img[i].height, hipMemcpyHostToDevice);
        imgPad[i] = HostPadMat(img[i], N, N);    // pad to use sliding windows
    }
    PrintData<<<1,1>>>();
    hipDeviceSynchronize();
    printf("%d\n", img[1].height);
    // test<<<20,20>>>();
    hipDeviceSynchronize();
    // double *a, *b, *d_B, *d_A, *d_C, *c;
    // hipMalloc((void**)&d_C, sizeof(double)*3*3);
    // hipMalloc((void**)&d_B, 6*sizeof(double));
    // hipMalloc((void**)&d_A, 6*sizeof(double));
    // c = new double[9];
    // b = new double[6];
    // a = new double[6];
    // double alpha = 1, beta = 0;
    // for(int i=0;i<6;i++)
    // {
    //     b[i] = i+1;
    //     a[i] = i+1;
    // }
    // hipMemcpy(d_A, a, 6*sizeof(double), hipMemcpyHostToDevice);
    // hipMemcpy(d_B, b, 6*sizeof(double), hipMemcpyHostToDevice);
    // hipblasHandle_t handle;
    // stat = hipblasCreate(&handle);
    // if (stat != HIPBLAS_STATUS_SUCCESS) {
    //     printf ("CUBLAS initialization failed\n");
    //     return EXIT_FAILURE;
    // }
    // hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 3, 3, 2, &alpha, d_A, 3, d_B, 2, &beta, d_C, 3);
    // hipMemcpy(c, d_C, sizeof(double)*9, hipMemcpyDeviceToHost);
    // printf("%f\n", c[1]);
    // hipblasDestroy(handle);
}